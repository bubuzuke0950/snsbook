#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hh.h"

#define N     ( 65536 )
#define I_EXT ( 9.0 )
#define NT    ( 100000 ) // 100000 * 0.01 = 1000 ms
#define DT    ( 0.01 ) // ms

#define BLOCK_SIZE ( 32 )

typedef struct {
  double *v, *m, *h, *n;
} hh_t;

__device__ double dev_alpha_m ( const double v ) { return ( 2.5 - 0.1 * v ) / ( exp ( 2.5 - 0.1 * v ) - 1.0 ); }
__device__ double dev_beta_m  ( const double v ) { return 4.0 * exp ( - v / 18.0 ); }
__device__ double dev_alpha_h ( const double v ) { return 0.07 * exp ( - v / 20.0 ); }
__device__ double dev_beta_h  ( const double v ) { return 1.0 / ( exp ( 3.0 - 0.1 * v ) + 1.0 ); }
__device__ double dev_alpha_n ( const double v ) { return ( 0.1 - 0.01 * v ) / ( exp ( 1 - 0.1 * v ) - 1.0 ); }
__device__ double dev_beta_n  ( const double v ) { return 0.125 * exp ( - v / 80.0 ); }

extern "C" { void timer_start ( void ); }
extern "C" { double timer_elapsed ( void ); }

static void initialize ( hh_t *neuron )
{
  hipMallocManaged ( &neuron -> v, N * sizeof ( double ) );
  hipMallocManaged ( &neuron -> m, N * sizeof ( double ) );
  hipMallocManaged ( &neuron -> h, N * sizeof ( double ) );
  hipMallocManaged ( &neuron -> n, N * sizeof ( double ) );

  double v = E_LEAK;
  for ( int32_t i = 0; i < N; i++ ) {
    neuron -> v [ i ] = v;
    neuron -> m [ i ] = inf_m ( v );
    neuron -> h [ i ] = inf_h ( v );
    neuron -> n [ i ] = inf_n ( v );
  }
}

static void finalize ( hh_t *neuron )
{
  hipFree ( neuron -> v );
  hipFree ( neuron -> m );
  hipFree ( neuron -> h );
  hipFree ( neuron -> n );
}

__global__ void kernel_gpu ( hh_t *neuron )
{
  int32_t i = threadIdx.x + blockIdx.x * blockDim.x;

  if ( i < N ) {
    double *v = &neuron -> v [ i ];
    double *m = &neuron -> m [ i ];
    double *h = &neuron -> h [ i ];
    double *n = &neuron -> n [ i ];

    double dv = DT * ( 1.0 / C ) * ( - G_LEAK * ( *v - E_LEAK )
				     - G_NA * *m * *m * *m * *h * ( *v - E_NA )
				     - G_K * *n * *n * *n * *n * ( *v - E_K )
				     + I_EXT );
    double dm = DT * ( dev_alpha_m ( *v ) * ( 1.0 - *m ) - dev_beta_m ( *v ) * *m );
    double dh = DT * ( dev_alpha_h ( *v ) * ( 1.0 - *h ) - dev_beta_h ( *v ) * *h );
    double dn = DT * ( dev_alpha_n ( *v ) * ( 1.0 - *n ) - dev_beta_n ( *v ) * *n );

    neuron -> v [ i ] += dv;
    neuron -> m [ i ] += dm;
    neuron -> h [ i ] += dh;
    neuron -> n [ i ] += dn;
  }
}

static void kernel_cpu ( hh_t *neuron )
{

  for ( int32_t i = 0; i < N; i++ ) {
    double *v = &neuron -> v [ i ];
    double *m = &neuron -> m [ i ];
    double *h = &neuron -> h [ i ];
    double *n = &neuron -> n [ i ];

    double dv = DT * ( 1.0 / C ) * ( - G_LEAK * ( *v - E_LEAK )
				     - G_NA * *m * *m * *m * *h * ( *v - E_NA )
				     - G_K * *n * *n * *n * *n * ( *v - E_K )
				     + I_EXT );
    double dm = DT * ( alpha_m ( *v ) * ( 1.0 - *m ) - beta_m ( *v ) * *m );
    double dh = DT * ( alpha_h ( *v ) * ( 1.0 - *h ) - beta_h ( *v ) * *h );
    double dn = DT * ( alpha_n ( *v ) * ( 1.0 - *n ) - beta_n ( *v ) * *n );

    *v += dv;
    *m += dm;
    *h += dh;
    *n += dn;
  }
}

int main ( void )
{
  hh_t *neuron;
  hipMallocManaged ( &neuron, sizeof ( hh_t ) );

  initialize ( neuron );

  int32_t gs = ( N + ( BLOCK_SIZE - 1) ) / BLOCK_SIZE;

  timer_start ();
  for ( int32_t nt = 0; nt < NT; nt++ ) {
    double t = DT * nt;
    //printf ( "%f %f\n", t, neuron -> v [ 0 ] );
    kernel_gpu <<< gs, BLOCK_SIZE >>> ( neuron );
    //kernel_cpu ( &neuron );
  }
  double elapsedTime = timer_elapsed ();
  printf ( "Elapsed time = %f sec.\n", elapsedTime);

  finalize ( neuron );
}
