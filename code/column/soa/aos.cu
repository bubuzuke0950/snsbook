#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hh.h"

#define N     ( 65536 )
#define I_EXT ( 9.0 )
#define NT    ( 100000 ) // 100000 * 0.01 = 1000 ms
#define DT    ( 0.01 ) // ms

#define BLOCK_SIZE ( 32 )

typedef struct {
  double v, m, h, n;
} hh_t;

__device__ double dev_alpha_m ( const double v ) { return ( 2.5 - 0.1 * v ) / ( exp ( 2.5 - 0.1 * v ) - 1.0 ); }
__device__ double dev_beta_m  ( const double v ) { return 4.0 * exp ( - v / 18.0 ); }
__device__ double dev_alpha_h ( const double v ) { return 0.07 * exp ( - v / 20.0 ); }
__device__ double dev_beta_h  ( const double v ) { return 1.0 / ( exp ( 3.0 - 0.1 * v ) + 1.0 ); }
__device__ double dev_alpha_n ( const double v ) { return ( 0.1 - 0.01 * v ) / ( exp ( 1 - 0.1 * v ) - 1.0 ); }
__device__ double dev_beta_n  ( const double v ) { return 0.125 * exp ( - v / 80.0 ); }

extern "C" { void timer_start ( void ); }
extern "C" { double timer_elapsed ( void ); }

static void initialize ( hh_t neuron [ ] )
{
  double v = E_LEAK;
  for ( int32_t i = 0; i < N; i++ ) {
    neuron [ i ] . v = v;
    neuron [ i ] . m = inf_m ( v );
    neuron [ i ] . h = inf_h ( v );
    neuron [ i ] . n = inf_n ( v );
  }
}

static void finalize ( hh_t neuron [ ] ) { hipFree ( neuron ); }

__global__ void kernel_gpu ( hh_t neuron [ ] )
{
  int32_t i = threadIdx.x + blockIdx.x * blockDim.x;

  if ( i < N ) {
    double *v = & neuron [ i ] . v;
    double *m = & neuron [ i ] . m;
    double *h = & neuron [ i ] . h;
    double *n = & neuron [ i ] . n;

    double dv = DT * ( 1.0 / C ) * ( - G_LEAK * ( *v - E_LEAK )
				     - G_NA * *m * *m * *m * *h * ( *v - E_NA )
				     - G_K * *n * *n * *n * *n * ( *v - E_K )
				     + I_EXT );
    double dm = DT * ( dev_alpha_m ( *v ) * ( 1.0 - *m ) - dev_beta_m ( *v ) * *m );
    double dh = DT * ( dev_alpha_h ( *v ) * ( 1.0 - *h ) - dev_beta_h ( *v ) * *h );
    double dn = DT * ( dev_alpha_n ( *v ) * ( 1.0 - *n ) - dev_beta_n ( *v ) * *n );

    *v += dv;
    *m += dm;
    *h += dh;
    *n += dn;
  }
}

static void kernel_cpu ( hh_t neuron [ ] )
{
  for ( int32_t i = 0; i < N; i++ ) {
    double *v = & neuron [ i ] . v;
    double *m = & neuron [ i ] . m;
    double *h = & neuron [ i ] . h;
    double *n = & neuron [ i ] . n;

    double dv = DT * ( 1.0 / C ) * ( - G_LEAK * ( *v - E_LEAK )
				     - G_NA * *m * *m * *m * *h * ( *v - E_NA )
				     - G_K * *n * *n * *n * *n * ( *v - E_K )
				     + I_EXT );
    double dm = DT * ( alpha_m ( *v ) * ( 1.0 - *m ) - beta_m ( *v ) * *m );
    double dh = DT * ( alpha_h ( *v ) * ( 1.0 - *h ) - beta_h ( *v ) * *h );
    double dn = DT * ( alpha_n ( *v ) * ( 1.0 - *n ) - beta_n ( *v ) * *n );

    *v += dv;
    *m += dm;
    *h += dh;
    *n += dn;
  }
}

int main ( void )
{
  hh_t *neuron;
  hipMallocManaged ( &neuron, N * sizeof ( hh_t ) );

  initialize ( neuron );

  int32_t gs = ( N + ( BLOCK_SIZE - 1) ) / BLOCK_SIZE;

  timer_start ();
  for ( int32_t nt = 0; nt < NT; nt++ ) {
    double t = DT * nt;
    //printf ( "%f %f\n", t, neuron [ 0 ] . v );
    kernel_gpu <<< gs, BLOCK_SIZE >>> ( neuron );
    //kernel_cpu ( neuron );
  }
  double elapsedTime = timer_elapsed ();
  printf ( "Elapsed time = %f sec.\n", elapsedTime);

  finalize ( neuron );
}
