#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hh.h"

double alpha_m ( const double v ) { return ( 2.5 - 0.1 * v ) / ( exp ( 2.5 - 0.1 * v ) - 1.0 ); }
double beta_m  ( const double v ) { return 4.0 * exp ( - v / 18.0 ); }
double alpha_h ( const double v ) { return 0.07 * exp ( - v / 20.0 ); }
double beta_h  ( const double v ) { return 1.0 / ( exp ( 3.0 - 0.1 * v ) + 1.0 ); }
double alpha_n ( const double v ) { return ( 0.1 - 0.01 * v ) / ( exp ( 1 - 0.1 * v ) - 1.0 ); }
double beta_n  ( const double v ) { return 0.125 * exp ( - v / 80.0 ); }

double inf_m ( const double v ) { return alpha_m ( v ) / ( alpha_m ( v ) + beta_m ( v ) ); }
double inf_h ( const double v ) { return alpha_h ( v ) / ( alpha_h ( v ) + beta_h ( v ) ); }
double inf_n ( const double v ) { return alpha_n ( v ) / ( alpha_n ( v ) + beta_n ( v ) ); }
