#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <stdint.h>
#include <stdbool.h>
#include <SFMT.h>

#define N       ( 4000 )            // # of neurons 
#define N_E     ( 3200 )            // # of excitatory neurons
#define N_I     ( ( N ) - ( N_E ) ) // # of inhibitory neurons 

#define T       ( 1000 ) // total simulation time ( 1000 ms, unused )
#define DT      ( 1.   ) // delta t = 1 ms
#define NT      ( 1000 ) // # of steps ( = T / DT )

#define TAU_M   (  20. ) // membrane time constant ( ms )
#define TAU_E   (  5.  ) // excitatory synapse time constant ( ms )
#define TAU_I   (  10. ) // inhibitory synapse time constant ( ms )

#define V_REST  ( -49. ) // resting potential ( mV )
#define V_INIT  ( -60. ) // initial value ( mV )
#define V_RESET ( -60. ) // reset potential ( mV )
#define THETA   ( -50. ) // threshold ( mV )

#define G_E     ( ( 0.1 * 1.62 ) / ( TAU_E ) ) // 1 / peak excitatory synaptic conductance
#define G_I     ( ( 0.1 * -9.  ) / ( TAU_I ) ) // 1 / peak inhibitory synaptic conductance
#define P       ( 0.2 ) // probability of making synaptic connections

typedef struct {
  float *v, *ge, *gi, *w;
  bool *s;
  int32_t *wc;
  int32_t nc;
  sfmt_t rng;
  FILE *file;
} network_t;

extern "C" { void timer_start ( void ); }
extern "C" { double timer_elapsed ( void ); }

#define BLOCK_SIZE ( 32 )
#define GRID_SIZE  ( ( ( N ) + ( BLOCK_SIZE ) - 1 ) / ( BLOCK_SIZE ) )
#define NTHSYN ( 2048 )

void initialize ( network_t *n )
{
  // PRNG
  sfmt_init_gen_rand ( &n -> rng, 23 );

  // File
  n -> file = fopen ( "spike.dat", "w");

  // Cell parameters
  hipMallocManaged ( &n -> v,  N * sizeof ( float ) );
  hipMallocManaged ( &n -> ge, N * sizeof ( float ) );
  hipMallocManaged ( &n -> gi, N * sizeof ( float ) );
  hipMallocManaged ( &n -> s,  N * sizeof ( bool ) );

  for ( int i = 0; i < N; i++ ) {
    n -> v [ i ] = V_INIT + 10. * sfmt_genrand_real2 ( &n -> rng );
    n -> ge [ i ] = 0.;
    n -> gi [ i ] = 0.;
    n -> s [ i ] = 0;
  }

  // Synaptic connections

  // Temporary building connections
  float *w  = ( float * ) calloc ( N * N, sizeof ( float ) );
  int32_t *wc = ( int32_t * ) calloc ( N * N, sizeof ( int32_t ) );
  int32_t maxcol = 0;
  for ( int32_t i = 0; i < N; i++ ) {
    int32_t col = 0;
    for ( int32_t j = 0; j < N; j++ ) {
      if ( sfmt_genrand_real2 ( &n -> rng ) < P ) {
	w [ j + N * i ] = 1.;
	wc [ col + N * i ] = j;
	col++;
      }
    }
    wc [ col + N * i ] = -1; // sentinel
    if ( maxcol < col ) { maxcol = col; }
  }

  // Building ELL matrix
  n -> nc = maxcol + 1; // +1 for sentinel
  hipMallocManaged ( &n -> w,  n -> nc * N * sizeof ( float ) );
  hipMallocManaged ( &n -> wc, n -> nc * N * sizeof ( int32_t ) );
  for ( int32_t i = 0; i < N; i++ ) {
    int32_t j = 0, k = 0;
    while ( ( k = wc [ j + N * i ] ) != -1 ) {
      n -> w  [ j + n -> nc * i ] = w [ k + N * i ];
      n -> wc [ j + n -> nc * i ] = k;
      j++;
    }
    while ( j < n -> nc ) {
      n -> w  [ j + n -> nc * i ] = 0; // filling 0
      n -> wc [ j + n -> nc * i ] = -1; // filling -1
      j++;
    }
  }

  free ( w );
  free ( wc );
}

void finalize ( network_t *n )
{
  hipFree ( n -> v );
  hipFree ( n -> ge );
  hipFree ( n -> gi );
  hipFree ( n -> s );
  hipFree ( n -> w );
  hipFree ( n -> wc );
  fclose ( n -> file );
}

__global__ void calculateSynapticInputs_a ( network_t *n )
{
  int32_t i = threadIdx.x + blockIdx.x * blockDim.x;

  if ( i < N ) {
    float re = 0, ri = 0;
    for ( int32_t j = 0, k = 0; ( k = n -> wc [ j + n -> nc * i ] ) != -1; j++ ){
      float r = n -> w [ j + n -> nc * i ] * n -> s [ k ];
      if ( k < N_E ) { re += r; } else { ri += r; } 
    }
    n -> ge [ i ] = exp ( - DT / TAU_E ) * n -> ge [ i ] + re;
    n -> gi [ i ] = exp ( - DT / TAU_I ) * n -> gi [ i ] + ri;
  }
}

__global__ void calculateSynapticInputs_b ( network_t *n )
{
  int32_t _i = threadIdx.x + blockIdx.x * blockDim.x;

  int32_t i = _i / NTHSYN;
  int32_t j = _i % NTHSYN;

  __shared__ float s_re [ NTHSYN ], s_ri [ NTHSYN ];

  if ( i < N ) {
    int32_t l = ( j < n -> nc ) ? n -> wc [ j + n -> nc * i] : -1;
    s_re [ j ] = ( l != -1 && l < N_E  ) ? n -> w [ j + n -> nc * i ] * n -> s [ l ] : 0.;
    s_ri [ j ] = ( l != -1 && l >= N_E ) ? n -> w [ j + n -> nc * i ] * n -> s [ l ] : 0.;
    for ( int32_t k = NTHSYN; k < n -> nc; k += NTHSYN ) {
      int32_t l = ( j + k < n -> nc ) ? n -> wc [ j + k + n -> nc * i] : -1;
      s_re [ j ] += ( l != -1 && l < N_E )  ? n -> w [ j + k + n -> nc * i ] * n -> s [ l ] : 0.;
      s_ri [ j ] += ( l != -1 && l >= N_E ) ? n -> w [ j + k + n -> nc * i ] * n -> s [ l ] : 0.;
    }
  }
  __syncthreads ( );

  for ( int32_t k = NTHSYN / 2; k > 0; k >>= 1 ) {
    if ( i < N && j < k ) {
      s_re [ j ] += s_re [ j + k ];
      s_ri [ j ] += s_ri [ j + k ];
    }
    __syncthreads ( );
  }

  if ( i < N && j == 0 ) {
    n -> ge [ i ] = exp ( - DT / TAU_E ) * n -> ge [ i ] + s_re [ 0 ];
    n -> gi [ i ] = exp ( - DT / TAU_I ) * n -> gi [ i ] + s_ri [ 0 ];
  }
}

__global__ void updateCellParameters ( network_t *n )
{
  int32_t i = threadIdx.x + blockIdx.x * blockDim.x;

  if ( i < N ) {
    n -> v [ i ] += DT * ( - ( n -> v [ i ] - V_REST ) + G_E * n -> ge [ i ] + G_I * n -> gi [ i ] ) / TAU_M;
    n -> s [ i ] = ( n -> v [ i ] > THETA );
    n -> v [ i ] = ( n -> s [ i ] ) * V_RESET + ( ! n -> s [ i ] ) * n -> v [ i ];
  }
}

void outputSpike ( const int32_t nt, network_t *n )
{
  for ( int32_t i = 0; i < N; i++ ) {
    if ( n -> s [ i ] ) { fprintf ( n -> file, "%f %d\n", DT * ( nt + 1 ), i ); } // Spike time is not t but t + DT
  }
}

void loop ( network_t *n )
{
  timer_start ( );

  for ( int32_t nt = 0; nt < NT; nt++ ) {
    calculateSynapticInputs_a <<< GRID_SIZE, BLOCK_SIZE >>> ( n );
    //calculateSynapticInputs_b <<< N, NTHSYN >>> ( n );
    updateCellParameters <<< GRID_SIZE, BLOCK_SIZE >>> ( n );
    hipDeviceSynchronize ( );
    outputSpike ( nt, n );
  }

  double elapsedTime = timer_elapsed ( );
  printf ( "Elapsed time = %f sec.\n", elapsedTime);
}

int main ( void )
{
  network_t *n;
  hipMallocManaged ( &n, sizeof ( network_t ) );

  initialize ( n );
  loop ( n );
  finalize ( n );

  hipFree ( n );
}
